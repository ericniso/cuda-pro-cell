#include "hip/hip_runtime.h"
#include <fstream>
#include <math.h>
#include <inttypes.h>
#include <map>
#include "simulation/data_types.h"
#include "io/parser.h"

namespace procell { namespace io
{

struct cell_type_comparator
{
    __host__
    __device__
    bool
    operator()(const simulation::cell_type& lhs, const simulation::cell_type& rhs)
    {
        return lhs.probability > rhs.probability;
    }
};

struct cell_type_reduce_binary :
    public thrust::binary_function<simulation::cell_type,
                                    simulation::cell_type,
                                    simulation::cell_type>
{
    __device__
    __host__
    simulation::cell_type
    operator()(simulation::cell_type& c1, simulation::cell_type& c2)
    {
        simulation::cell_type t =
        {
            .name = 0,
            .probability = c1.probability + c2.probability,
            .timer = 0.0,
            .sigma = 0.0
        };

        return t;
    }

};

__host__
void
assert_probability_sum(simulation::cell_types& h_params)
{
    simulation::cell_type base =
    {
        .name = 0,
        .probability = 0.0,
        .timer = 0.0,
        .sigma = 0.0
    };

    simulation::cell_type result =
        thrust::reduce(h_params.begin(), h_params.end(),
                        base, cell_type_reduce_binary());

    double_t err = 1 / pow(10.0, 15.0);
    if (abs(1.0 - result.probability) > err)
    {
        std::cout << "ERROR: probability distribution of cell types does not sum to 1, aborting." << std::endl;
        exit(EXIT_FAILURE);
    }
}

__host__
void
load_fluorescences(char* histogram, simulation::fluorescences& data,
                    simulation::initial_bounds& bounds,
                    simulation::fluorescences& predicted_values,
                    double_t threshold,
                    uint64_t* size)
{
    simulation::host_map_results m_results;
    uint64_t total = 0;
    std::ifstream in(histogram);

    bool first = true;
    double_t value = 0.0;
    uint64_t frequency = 0;
    simulation::fluorescence previous;
    uint64_t previous_start = 0;
    while (in >> value >> frequency)
    {
        if (frequency == 0)
            continue;

        uint64_t start_index = 0;
        if (!first)
        {
            start_index = previous_start + previous.frequency;
        }
        first = false;

        total += frequency;
        simulation::fluorescence f =
        {
            .value = value,
            .frequency = frequency
        };
        previous = f;
        previous_start = start_index;
        data.push_back(f);
        bounds.push_back(start_index);

        double_t curr_val = value;
        while (curr_val >= threshold)
        {
            simulation::host_map_results::iterator it
                = m_results.find(curr_val);
            if (it == m_results.end())
            {
                m_results.insert(std::make_pair(curr_val, 0));
            }

            curr_val = curr_val / 2;
        }
    }

    in.close();

    for (simulation::host_map_results::iterator it = m_results.begin();
        it != m_results.end(); it++)
    {
        simulation::fluorescence f = 
        {
            .value = it->first,
            .frequency = 0
        };
        predicted_values.push_back(f);
    }

    *size = total;
}

__host__
void
load_cell_types(char* types, simulation::cell_types& data)
{
    std::ifstream in(types);

    int32_t name = 0;
    double_t probability = 0.0;
    double_t timer = 0.0;
    double_t sigma = 0.0;

    while (in >> probability >> timer >> sigma)
    {
        simulation::cell_type c =
            simulation::create_cell_type(name, probability, timer, sigma);

        data.push_back(c);
        
        name++;
    }

    in.close();

    assert_probability_sum(data);

    simulation::cell_type* d_params = NULL;
    hipMalloc((void**) &d_params, data.size() * sizeof(simulation::cell_type));
    
    thrust::sort(data.begin(), data.end(), cell_type_comparator());
}

__host__
bool
save_fluorescences(char* filename, 
                    simulation::fluorescences& results)
{
    std::ofstream out(filename);

    if (!out.is_open())
        return false;

    out.precision(10);

    for (uint64_t i = 0; i < results.size(); i++)
    {
        if (results[i].frequency > 0)
        {
            out << results[i].value << "\t"
                << results[i].frequency << std::endl;
        }
    }

    out.close();

    return true;
}

} // End io namespace
    
} // End procell namespace
