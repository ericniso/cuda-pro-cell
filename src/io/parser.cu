#include "hip/hip_runtime.h"
#include <fstream>
#include <math.h>
#include <inttypes.h>
#include <map>
#include <thrust/sort.h>
#include "simulation/data_types.h"
#include "io/parser.h"

namespace procell { namespace io
{

struct cell_type_comparator
{
    __host__
    __device__
    bool
    operator()(const simulation::cell_type& lhs, const simulation::cell_type& rhs)
    {
        return lhs.proportion > rhs.proportion;
    }
};

struct cell_type_reduce_binary :
    public thrust::binary_function<simulation::cell_type,
                                    simulation::cell_type,
                                    simulation::cell_type>
{
    __device__
    __host__
    simulation::cell_type
    operator()(simulation::cell_type& c1, simulation::cell_type& c2)
    {
        simulation::cell_type t =
        {
            .name = 0,
            .proportion = c1.proportion + c2.proportion,
            .timer = 0.0,
            .sigma = 0.0
        };

        return t;
    }

};

__host__
void
assert_proportion_sum(simulation::cell_types& h_params)
{
    simulation::cell_type base =
    {
        .name = 0,
        .proportion = 0.0,
        .timer = 0.0,
        .sigma = 0.0
    };

    simulation::cell_type result =
        thrust::reduce(h_params.begin(), h_params.end(),
                        base, cell_type_reduce_binary());

    double_t err = 1 / pow(10.0, 15.0);
    if (abs(1.0 - result.proportion) > err)
    {
        std::cout << "ERROR: proportion distribution of cell types does not sum to 1, aborting." << std::endl;
        exit(EXIT_FAILURE);
    }
}

__host__
void
load_fluorescences(const char* histogram, simulation::fluorescences& data,
                    simulation::initial_bounds& bounds,
                    simulation::fluorescences_result& predicted_values,
                    double_t& threshold,
                    uint64_t* size)
{
    simulation::host_map_results m_results;
    uint64_t total = 0;
    std::ifstream in(histogram);

    if (threshold == 0.0)
    {
        double_t value = 0.0;
        uint64_t frequency = 0;

        while (in >> value >> frequency)
        {
            if (frequency > 0)
            {
                if (threshold == 0.0 || value < threshold)
                    threshold = value;
            }
        }

        in.clear();
        in.seekg(0);
    }

    bool first = true;
    double_t value = 0.0;
    uint64_t frequency = 0;
    simulation::fluorescence previous;
    uint64_t previous_start = 0;
    while (in >> value >> frequency)
    {
        if (frequency == 0)
            continue;

        uint64_t start_index = 0;
        if (!first)
        {
            start_index = previous_start + previous.frequency;
        }
        first = false;

        total += frequency;
        simulation::fluorescence f =
        {
            .value = value,
            .frequency = frequency
        };
        previous = f;
        previous_start = start_index;
        data.push_back(f);
        bounds.push_back(start_index);

        double_t curr_val = value;
        while (curr_val >= threshold)
        {
            simulation::host_map_results::iterator it
                = m_results.find(curr_val);
            if (it == m_results.end())
            {
                m_results.insert(std::make_pair(curr_val, 0));
            }

            curr_val = curr_val / 2;
        }
    }

    in.close();

    for (simulation::host_map_results::iterator it = m_results.begin();
        it != m_results.end(); it++)
    {
        simulation::fluorescence_with_ratio f = 
        {
            .value = it->first,
            .frequency = 0,
            .ratio = NULL
        };
        predicted_values.push_back(f);
    }

    *size = total;
}

__host__
void
load_cell_types(const char* types, simulation::cell_types& data)
{
    std::ifstream in(types);

    int32_t name = 0;
    double_t proportion = 0.0;
    double_t timer = 0.0;
    double_t sigma = 0.0;

    while (in >> proportion >> timer >> sigma)
    {
        simulation::cell_type c =
            simulation::create_cell_type(name, proportion, timer, sigma);

        data.push_back(c);
        
        name++;
    }

    in.close();

    assert_proportion_sum(data);

    simulation::cell_type* d_params = NULL;
    hipMalloc((void**) &d_params, data.size() * sizeof(simulation::cell_type));
    
    thrust::sort(data.begin(), data.end(), cell_type_comparator());
}

__host__
bool
save_fluorescences(std::ostream& stream,
                    bool save_ratio,
                    int32_t ratio_size,
                    simulation::fluorescences_result& results)
{

    stream.precision(10);

    for (uint64_t i = 0; i < results.size(); i++)
    {
        if (results[i].frequency > 0)
        {
            stream << results[i].value << "\t"
                << results[i].frequency;

            if (save_ratio)
            {
                for (int32_t j = 0; j < ratio_size; j++)
                {
                    stream << "\t" << results[i].ratio[j];
                }
            }

            stream << std::endl;
        }
    }

    return true;
}

} // End io namespace
    
} // End procell namespace
