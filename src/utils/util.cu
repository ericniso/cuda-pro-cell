#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils/util.h"

namespace procell { namespace utils
{
    
namespace device
{

__device__
hiprandState_t
init_random(uint64_t seed)
{
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);

    return state;
}

__device__
double_t
uniform_random(uint64_t seed)
{
    hiprandState_t state = init_random(seed);
    return hiprand_uniform_double(&state);
}

__device__
double_t
normal_random(uint64_t seed, double_t mean, double_t sd)
{
    hiprandState_t state = init_random(seed);
    return hiprand_log_normal_double(&state, mean, sd);
}

} // End device namespace
    
} // End utils namespace
    
} // End procell namespace
