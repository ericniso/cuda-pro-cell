#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils/util.h"

namespace procell { namespace utils
{
    
namespace device
{

__device__
hiprandState_t
init_random(uint64_t seed)
{
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);

    return state;
}

__device__
double_t
uniform_random(uint64_t seed)
{
    hiprandState_t state = init_random(seed);
    return hiprand_uniform_double(&state);
}

__device__
double_t
normal_random(uint64_t seed)
{
    hiprandState_t state = init_random(seed);
    return hiprand_normal_double(&state);
}

} // End device namespace
    
} // End utils namespace
    
} // End procell namespace
