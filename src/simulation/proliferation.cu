#include "hip/hip_runtime.h"
#include <iostream>
#include <inttypes.h>
#include <time.h>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <map>
#include "simulation/proliferation.h"
#include "simulation/cell.h"
#include "simulation/data_types.h"
#include "utils/util.h"

#define MAX_SYNC_DEPTH (24)

#define REMOVE 0
#define ALIVE 1
#define INACTIVE 2

namespace procell { namespace simulation
{

__host__
bool
proliferate(simulation::cell_types& h_params,
            uint64_t size, cell* h_cells, double_t t_max, double_t threshold,
            host_map_results& m_results)
{
    device::cell_types d_params = h_params;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0 /* TODO check devices number */);
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_SYNC_DEPTH);

    cell* h_active_cells = h_cells;
    cell* d_current_stage = NULL;
    uint64_t new_size = size;
    hipMalloc((void**) &d_current_stage, new_size * sizeof(cell));
    hipMemcpy(d_current_stage, h_active_cells, new_size * sizeof(cell),
        hipMemcpyHostToDevice);

    uint64_t divisions = 0;
    while (new_size > 0)
    {
        uint64_t depth = utils::max_recursion_depth(new_size);

        // Check if GPU has enough memory to compute next stage
        if (depth == 0)
        {
            std::cout << "--- ERROR: out of GPU memory" << std::endl;
            std::cout << "--- Total iterations: " << divisions << std::endl;
            std::cout << "--- Copying partial results to file...";
            std::cout << "copied, aborting." << std::endl;
            return false;
        }

        run_iteration(d_params,
            t_max,
            threshold,
            prop.maxThreadsPerBlock,
            &d_current_stage,
            new_size,
            depth);

        new_size = new_size * pow(2, depth);
        new_size = count_future_proliferation_events(
            &d_current_stage, new_size, m_results);

        divisions++;
    }

    return true;
}

__host__
void
run_iteration(device::cell_types& d_params, double_t t_max, double_t threshold,
    uint32_t max_threads_per_block, cell** d_current_stage,
    uint64_t& current_size, uint64_t depth)
{
    host_tree_levels h_tree_levels;
    h_tree_levels.push_back(*d_current_stage);

    for (uint8_t i = 1; i < (depth + 1); i++)
    {
        cell* level_population = NULL;
        uint32_t cell_level_size = current_size * pow(2, i);
        hipMalloc((void**) &level_population,
            cell_level_size * sizeof(cell));

        h_tree_levels.push_back(level_population);
    }

    device::device_tree_levels d_tree_levels = h_tree_levels;

    uint64_t random_seed = time(NULL);

    uint64_t original_size = current_size;
    uint16_t n_blocks = round(0.5 + current_size / max_threads_per_block);

    device::proliferate<<<n_blocks, max_threads_per_block>>>
        (thrust::raw_pointer_cast(d_params.data()), d_params.size(),
        original_size,
        thrust::raw_pointer_cast(d_tree_levels.data()),
        threshold,
        t_max,
        random_seed,
        depth,
        0,
        0);

    hipDeviceSynchronize();

    for (uint8_t i = 0; i < depth; i++)
    {
        hipFree(h_tree_levels[i]);
    }

    *d_current_stage = h_tree_levels[depth];
}

__host__
uint64_t
count_future_proliferation_events(cell** d_stage, uint64_t size,
    host_map_results& m_results)
{
    host_cells new_stage;
    cell* h_stage = (cell*) malloc(size * sizeof(cell));
    hipMemcpy(h_stage, *d_stage, size * sizeof(cell), hipMemcpyDeviceToHost);

    for (uint64_t i = 0; i < size; i++)
    {
        switch (h_stage[i].state)
        {
            case INACTIVE:
            {
                double_t fluorescence = h_stage[i].fluorescence;
                host_map_results::iterator it
                    = m_results.find(fluorescence);
                if (it == m_results.end())
                {
                    m_results.insert(std::make_pair(fluorescence, 1));
                }
                else
                {
                    it->second = it->second + 1;
                }
            }
            break;

            case ALIVE:
            {
                new_stage.push_back(h_stage[i]);
            }
            break;

            case REMOVE:
            {
                // Do nothing
            }
            break;
        }
    }

    uint64_t new_size = new_stage.size();
    hipMalloc((void**) d_stage, new_size * sizeof(cell));
    hipMemcpy(*d_stage, thrust::raw_pointer_cast(new_stage.data()),
        new_size * sizeof(cell), hipMemcpyHostToDevice);
    new_stage.clear();
    new_stage.shrink_to_fit();

    free(h_stage);

    return new_size;
}

namespace device
{
    
__global__
void
proliferate(cell_type* d_params, uint64_t size,
            uint64_t original_size,
            cell** cell_tree_levels,
            double_t fluorescence_threshold,
            double_t t_max,
            uint64_t seed,
            uint64_t depth,
            uint64_t current_depth,
            uint64_t offset)
{

    __shared__ bool proliferation;

    uint64_t id = offset + threadIdx.x + blockIdx.x * blockDim.x;

    if (id < original_size)
    {
        uint64_t next_id = id * 2;
        uint64_t next_depth = current_depth + 1;
        cell current = cell_tree_levels[current_depth][id];

        if (current_depth < depth)
        {
            if (current_depth > 0 && cell_tree_levels[current_depth][id].state != ALIVE)
            {
                if (cell_tree_levels[current_depth][id].state == INACTIVE)
                {
                    cell_tree_levels[next_depth][next_id] = current;
                    cell_tree_levels[next_depth][next_id + 1].state = REMOVE;
                }
                else
                {
                    cell_tree_levels[next_depth][next_id].state = REMOVE;
                    cell_tree_levels[next_depth][next_id + 1].state = REMOVE;
                }
            }
            else if (!cell_will_divide(current, fluorescence_threshold, t_max))
            {
                current.state = INACTIVE;
                cell_tree_levels[next_depth][next_id] = current;
                cell_tree_levels[next_depth][next_id + 1].state = REMOVE;
            }
            else
            {
                double_t fluorescence = current.fluorescence / 2;
                int32_t type = current.type;
                double_t t = current.t + current.timer;

                // Differentiate seeds
                uint64_t seed_c1 = seed + current.timer * 10000 + id;
                uint64_t seed_c2 = seed - current.timer * 10000 + id;

                cell c1 = create_cell(d_params, size, seed_c1,
                    type, fluorescence, t);

                cell c2 = create_cell(d_params, size, seed_c2,
                    type, fluorescence, t);
                
                c1.state = ALIVE;
                c2.state = ALIVE;

                cell_tree_levels[next_depth][next_id] = c1;
                cell_tree_levels[next_depth][next_id + 1] = c2;

                proliferation = true;
            }

            if (threadIdx.x == 0)
            {
                __syncthreads();

                uint64_t next_offset = id * 2;

                
                if (((current_depth + 1) == depth) || proliferation)
                {
                    proliferate<<<2, blockDim.x>>>(d_params, size,
                        original_size * 2,
                        cell_tree_levels,
                        fluorescence_threshold, t_max, seed,
                        depth, next_depth,
                        next_offset);
                }
                else
                {
                    uint64_t last_level_size = pow(2, depth);
                    apply_bounding<<<2, blockDim.x>>>(last_level_size,
                            cell_tree_levels,
                            depth,
                            next_depth,
                            next_offset);
                }
            }
        }
    }

}

__global__
void
apply_bounding(uint64_t original_size,
                cell** cell_tree_levels,
                uint64_t depth,
                uint64_t current_depth,
                uint64_t offset)
{
    uint64_t id = offset + threadIdx.x + blockIdx.x * blockDim.x;

    if (id < original_size)
    {
        cell current = cell_tree_levels[current_depth][id];
        if (current.state == INACTIVE)
        {
            uint64_t final_index = id * pow(2, depth - current_depth);
            cell_tree_levels[depth][final_index] = current;
        }
    }
}

__device__
bool
cell_will_divide(cell& c, double_t fluorescence_threshold, double_t t_max)
{
    return (c.timer > 0.0) && 
        (c.t + c.timer < t_max) &&
        (c.fluorescence / 2 > fluorescence_threshold);
}
    
} // End device namespace
    
} // End simulation namespace
    
} // End procell namespace
