#include "hip/hip_runtime.h"
#include <iostream>
#include <inttypes.h>
#include <time.h>
#include <math.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include "simulation/proliferation.h"
#include "simulation/cell.h"
#include "simulation/data_types.h"
#include "utils/util.h"

#define MAX_SYNC_DEPTH (24)

#define INACTIVE 0
#define ALIVE 1
#define REMOVE 2

namespace procell { namespace simulation
{

__host__
bool
proliferate(simulation::cell_types& h_params,
            uint64_t size, cell* h_cells, double_t t_max, double_t threshold,
            host_histogram_values& result_values,
            host_histogram_counts& result_counts)
{

    device::device_histogram_values d_result_values;
    device::device_histogram_counts d_result_counts;

    device::cell_types d_params = h_params;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0 /* TODO check devices number */);
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_SYNC_DEPTH);

    cell* h_active_cells = h_cells;
    cell* d_current_stage = NULL;
    proliferation_event* d_future_proliferation_events = NULL;
    proliferation_event_gap* d_future_proliferation_event_gaps = NULL;
    uint64_t new_size = size;
    hipMalloc((void**) &d_current_stage, new_size * sizeof(cell));
    hipMemcpy(d_current_stage, h_active_cells, new_size * sizeof(cell),
        hipMemcpyHostToDevice);

    uint64_t divisions = 0;
    while (new_size > 0)
    {
        uint64_t depth = utils::max_recursion_depth(new_size);

        // Check if GPU has enough memory to compute next stage
        if (depth == 0)
        {
            std::cout << "--- ERROR: out of GPU memory" << std::endl;
            std::cout << "--- Total iterations: " << divisions << std::endl;
            std::cout << "--- Copying partial results to file...";
            copy_result(result_values, result_counts,
                d_result_values, d_result_counts);
            std::cout << "copied, aborting." << std::endl;
            return false;
        }

        run_iteration(d_params,
            t_max,
            threshold,
            prop.maxThreadsPerBlock,
            &d_current_stage,
            &d_future_proliferation_events,
            &d_future_proliferation_event_gaps,
            new_size,
            depth);

        new_size = new_size * pow(2, depth);
        new_size = count_future_proliferation_events(
            &d_current_stage, d_future_proliferation_events, new_size,
            d_result_values, d_result_counts);

        hipFree(d_future_proliferation_events);

        divisions++;
    }

    hipFree(d_current_stage);

    copy_result(result_values, result_counts, d_result_values, d_result_counts);

    return true;
}

__host__
void
run_iteration(device::cell_types& d_params, double_t t_max, double_t threshold,
    uint32_t max_threads_per_block, cell** d_current_stage,
    proliferation_event** d_future_proliferation_events,
    proliferation_event_gap** d_final_proliferation_event_gaps,
    uint64_t& current_size, uint64_t depth)
{
    host_tree_levels h_tree_levels;
    host_event_tree_levels h_event_tree_levels;
    h_tree_levels.push_back(*d_current_stage);

    for (uint8_t i = 0; i < (depth + 1); i++)
    {
        if (i > 0)
        {
            cell* level_population = NULL;
            uint32_t cell_level_size = current_size * pow(2, i);
            hipMalloc((void**) &level_population,
                cell_level_size * sizeof(cell));

            h_tree_levels.push_back(level_population);
        }

        proliferation_event* level_events = NULL;
        uint32_t event_level_size = current_size * pow(2, i);
        hipMalloc((void**) &level_events,
            event_level_size * sizeof(proliferation_event));

        h_event_tree_levels.push_back(level_events);
    }

    device::device_tree_levels d_tree_levels = h_tree_levels;
    device::device_event_tree_levels d_event_tree_levels = h_event_tree_levels;

    uint64_t random_seed = time(NULL);

    uint64_t original_size = current_size;
    uint16_t n_blocks = round(0.5 + current_size / max_threads_per_block);
    current_size = current_size * pow(2, 1);
    
    hipMalloc((void**) d_future_proliferation_events,
        current_size * sizeof(proliferation_event));

    hipMalloc((void**) d_final_proliferation_event_gaps,
        current_size * sizeof(proliferation_event_gap));

    device::proliferate<<<n_blocks, max_threads_per_block>>>
        (thrust::raw_pointer_cast(d_params.data()), d_params.size(),
        original_size,
        thrust::raw_pointer_cast(d_tree_levels.data()),
        thrust::raw_pointer_cast(d_event_tree_levels.data()),
        *d_final_proliferation_event_gaps,
        threshold,
        t_max,
        random_seed,
        depth,
        0,
        0);

    hipDeviceSynchronize();

    for (uint8_t i = 0; i < depth; i++)
    {
        hipFree(h_tree_levels[i]);
        hipFree(h_event_tree_levels[i]);
    }

    *d_current_stage = h_tree_levels[depth];
    *d_future_proliferation_events = h_event_tree_levels[depth];
}

__host__
void
copy_result(host_histogram_values& result_values,
            host_histogram_counts& result_counts,
            device::device_histogram_values& partial_result_values,
            device::device_histogram_counts& partial_result_counts)
{
    thrust::sort_by_key(partial_result_values.begin(), partial_result_values.end(),
        partial_result_counts.begin());

    uint64_t result_values_size = partial_result_values.size();
    uint64_t result_counts_size = partial_result_counts.size();
    double_t* result_values_arr = (double_t*)
        malloc(result_values_size * sizeof(double_t));
    uint64_t* result_counts_arr = (uint64_t*)
        malloc(result_counts_size * sizeof(uint64_t));

    hipMemcpy(result_values_arr,
        thrust::raw_pointer_cast(partial_result_values.data()),
        result_values_size * sizeof(double_t),
        hipMemcpyDeviceToHost);

    hipMemcpy(result_counts_arr,
        thrust::raw_pointer_cast(partial_result_counts.data()),
        result_counts_size * sizeof(uint64_t),
        hipMemcpyDeviceToHost);

    result_values = host_histogram_values(result_values_arr,
        result_values_arr + result_values_size);
    result_counts = host_histogram_counts(result_counts_arr,
        result_counts_arr + result_counts_size);
}

__host__
uint64_t
count_future_proliferation_events(cell** d_stage, proliferation_event* d_events,
    uint64_t size,
    device::device_histogram_values& result_values,
    device::device_histogram_counts& result_counts)
{
    host_fluorescences result_stage;
    host_cells new_stage;
    proliferation_event* h_events = (proliferation_event*) malloc(size * sizeof(proliferation_event));
    cell* h_stage = (cell*) malloc(size * sizeof(cell));
    hipMemcpy(h_events, d_events, size * sizeof(proliferation_event), hipMemcpyDeviceToHost);
    hipMemcpy(h_stage, *d_stage, size * sizeof(cell), hipMemcpyDeviceToHost);

    for (uint64_t i = 0; i < size; i++)
    {
        switch (h_events[i])
        {
            case INACTIVE:
            {
                result_stage.push_back(h_stage[i].fluorescence);
            }
            break;

            case ALIVE:
            {
                new_stage.push_back(h_stage[i]);
            }
            break;

            case REMOVE:
            {
                // Do nothing
            }
            break;
        }
    }
    
    update_results(result_values, result_counts, result_stage);

    uint64_t new_size = new_stage.size();
    hipMalloc((void**) d_stage, new_size * sizeof(cell));
    hipMemcpy(*d_stage, thrust::raw_pointer_cast(new_stage.data()),
        new_size * sizeof(cell), hipMemcpyHostToDevice);
    new_stage.clear();
    new_stage.shrink_to_fit();

    free(h_stage);
    free(h_events);

    return new_size;
}

__host__
void
update_results(device::device_histogram_values& result_values,
                device::device_histogram_counts& result_counts,
                host_fluorescences& result_stage)
{
    uint64_t size = result_stage.size();
    double_t* d_fluorescence_values = NULL;
    hipMalloc((void**) &d_fluorescence_values,
        size * sizeof(double_t));

    hipMemcpy(d_fluorescence_values,
        thrust::raw_pointer_cast(result_stage.data()),
        size * sizeof(double_t),
        hipMemcpyHostToDevice);
    
    device::device_fluorescences d_fluorescences(d_fluorescence_values,
        d_fluorescence_values + size);
    
    // Calculate histogram
    thrust::sort(d_fluorescences.begin(), d_fluorescences.end());
    uint64_t num_bins = thrust::inner_product(d_fluorescences.begin(),
                            d_fluorescences.end() - 1,
                            d_fluorescences.begin() + 1,
                            (uint64_t) 1,
                            thrust::plus<uint64_t>(),
                            thrust::not_equal_to<double_t>());

    device::device_histogram_values new_histogram_values(num_bins);
    device::device_histogram_counts new_histogram_counts(num_bins);
    thrust::reduce_by_key(d_fluorescences.begin(), d_fluorescences.end(),
                    thrust::constant_iterator<uint64_t>(1),
                    new_histogram_values.begin(),
                    new_histogram_counts.begin());

    merge_histograms(result_values, result_counts,
        new_histogram_values, new_histogram_counts);

    d_fluorescences.clear();
    d_fluorescences.shrink_to_fit();
    hipFree(d_fluorescence_values);
}

__host__
void
merge_histograms(device::device_histogram_values& result_values,
                device::device_histogram_counts& result_counts,
                device::device_histogram_values& new_result_values,
                device::device_histogram_counts& new_result_counts)
{
    uint64_t result_size = result_values.size();
    uint64_t new_result_size = new_result_values.size();

    double_t* h_result_values =
        (double_t*) malloc(result_size * sizeof(double_t));
    uint64_t* h_result_counts =
        (uint64_t*) malloc(result_size * sizeof(uint64_t));
    double_t* h_new_result_values =
        (double_t*) malloc(new_result_size * sizeof(double_t));
    uint64_t* h_new_result_counts =
        (uint64_t*) malloc(new_result_size * sizeof(uint64_t));

    hipMemcpy(h_result_values,
        thrust::raw_pointer_cast(result_values.data()),
        result_size * sizeof(double_t),
        hipMemcpyDeviceToHost);
    hipMemcpy(h_result_counts,
        thrust::raw_pointer_cast(result_counts.data()),
        result_size * sizeof(uint64_t),
        hipMemcpyDeviceToHost);
    hipMemcpy(h_new_result_values,
        thrust::raw_pointer_cast(new_result_values.data()),
        new_result_size * sizeof(double_t),
        hipMemcpyDeviceToHost);
    hipMemcpy(h_new_result_counts,
        thrust::raw_pointer_cast(new_result_counts.data()),
        new_result_size * sizeof(uint64_t),
        hipMemcpyDeviceToHost);

    result_values.clear();
    result_values.shrink_to_fit();
    result_counts.clear();
    result_counts.shrink_to_fit();
    new_result_values.clear();
    new_result_values.shrink_to_fit();
    new_result_counts.clear();
    new_result_counts.shrink_to_fit();
    
    host_histogram_values values_to_add;
    host_histogram_counts counts_to_add;

    for (uint64_t i = 0; i < new_result_size; i++)
    {
        bool found = false;

        for (uint64_t j = 0; j < result_size; j++)
        {
            if (h_new_result_values[i] == h_result_values[j])
            {
                found = true;
                h_result_counts[j] += h_new_result_counts[i];
                break;
            }
        }

        if (!found)
        {
            values_to_add.push_back(h_new_result_values[i]);
            counts_to_add.push_back(h_new_result_counts[i]);
        }
    }

    double_t* d_result_values = NULL;
    uint64_t* d_result_counts = NULL;
    hipMalloc((void**) &d_result_values,
        (values_to_add.size() + result_size) * sizeof(double_t));
    hipMalloc((void**) &d_result_counts,
        (counts_to_add.size() + result_size) * sizeof(uint64_t));

    hipMemcpy(d_result_values,
        h_result_values,
        result_size * sizeof(double_t),
        hipMemcpyHostToDevice);
    hipMemcpy(&d_result_values[result_size],
        thrust::raw_pointer_cast(values_to_add.data()),
        values_to_add.size() * sizeof(double_t),
        hipMemcpyHostToDevice);
    hipMemcpy(d_result_counts,
        h_result_counts,
        result_size * sizeof(uint64_t),
        hipMemcpyHostToDevice);
    hipMemcpy(&d_result_counts[result_size],
        thrust::raw_pointer_cast(counts_to_add.data()),
        counts_to_add.size() * sizeof(uint64_t),
        hipMemcpyHostToDevice);

    result_values = device::device_histogram_values(
        d_result_values, d_result_values + (values_to_add.size() + result_size));
    result_counts = device::device_histogram_counts(
        d_result_counts, d_result_counts + (counts_to_add.size() + result_size));
    
    values_to_add.clear();
    values_to_add.shrink_to_fit();
    counts_to_add.clear();
    counts_to_add.shrink_to_fit();

    free(h_result_values);
    free(h_result_counts);
    free(h_new_result_values);
    free(h_new_result_counts);
}

namespace device
{
    
__global__
void
proliferate(cell_type* d_params, uint64_t size,
            uint64_t original_size,
            cell** cell_tree_levels,
            proliferation_event** event_tree_levels,
            proliferation_event_gap* proliferation_event_gaps,
            double_t fluorescence_threshold,
            double_t t_max,
            uint64_t seed,
            uint64_t depth,
            uint64_t current_depth,
            uint64_t offset)
{
    uint64_t id = offset + threadIdx.x + blockIdx.x * blockDim.x;

    if (id < original_size)
    {
        uint64_t next_id = id * 2;
        uint64_t next_depth = current_depth + 1;
        cell current = cell_tree_levels[current_depth][id];

        if (current_depth < depth)
        {
            if (current_depth > 0 && event_tree_levels[current_depth][id] != ALIVE)
            {
                if (event_tree_levels[current_depth][id] == INACTIVE)
                {
                    cell_tree_levels[next_depth][next_id] = current;

                    event_tree_levels[next_depth][next_id] = INACTIVE;
                    event_tree_levels[next_depth][next_id + 1] = REMOVE;
                }
                else
                {
                    event_tree_levels[next_depth][next_id] = REMOVE;
                    event_tree_levels[next_depth][next_id + 1] = REMOVE;
                }
            }
            else if (!cell_will_divide(current, fluorescence_threshold, t_max))
            {
                cell_tree_levels[next_depth][next_id] = current;

                event_tree_levels[next_depth][next_id] = INACTIVE;
                event_tree_levels[next_depth][next_id + 1] = REMOVE;
            }
            else
            {
                double_t fluorescence = current.fluorescence / 2;
                int32_t type = current.type;
                double_t t = current.t + current.timer;

                // Differentiate seeds
                uint64_t seed_c1 = seed + current.timer * 10000 + id;
                uint64_t seed_c2 = seed - current.timer * 10000 + id;

                cell c1 = create_cell(d_params, size, seed_c1,
                    type, fluorescence, t);

                cell c2 = create_cell(d_params, size, seed_c2,
                    type, fluorescence, t);

                cell_tree_levels[next_depth][next_id] = c1;
                cell_tree_levels[next_depth][next_id + 1] = c2;

                event_tree_levels[next_depth][next_id] = ALIVE;
                event_tree_levels[next_depth][next_id + 1] = ALIVE;
            }
            
            __syncthreads();

            if (threadIdx.x == 0)
            {
                uint64_t next_offset = id * 2;
                proliferate<<<2, blockDim.x>>>(d_params, size,
                    original_size * 2,
                    cell_tree_levels,
                    event_tree_levels,
                    proliferation_event_gaps,
                    fluorescence_threshold, t_max, seed,
                    depth, next_depth,
                    next_offset);
            }
        }
    }

}

__device__
bool
cell_will_divide(cell& c, double_t fluorescence_threshold, double_t t_max)
{
    return (c.timer > 0.0) && 
        (c.t + c.timer < t_max) &&
        (c.fluorescence / 2 > fluorescence_threshold);
}
    
} // End device namespace
    
} // End simulation namespace
    
} // End procell namespace
