#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <inttypes.h>
#include <math.h>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "simulation/data_types.h"
#include "simulation/cell.h"
#include "simulation/cells_population.h"
#include "utils/util.h"

namespace procell { namespace simulation
{

__host__
void
create_cells_population(cell_type* d_params, uint64_t params_size,
                        uint64_t initial_size,
                        fluorescences& h_input, initial_bounds& h_bounds,
                        cell* h_cells)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0 /* TODO check devices number */);
    
    uint16_t n_threads_per_block = prop.maxThreadsPerBlock;
    uint16_t n_blocks = round(0.5 + h_input.size() / n_threads_per_block);

    uint64_t bytes = initial_size * sizeof(cell);

    uint64_t random_seed = time(NULL);

    cell* d_cells = NULL;
    fluorescence* d_fluorescences = NULL;
    uint64_t* d_bounds = NULL;

    hipMalloc((void**) &d_cells, bytes);
    hipMalloc((void**) &d_fluorescences, h_input.size() * sizeof(fluorescence));
    hipMalloc((void**) &d_bounds, h_input.size() * sizeof(uint64_t));
    
    thrust::copy(h_input.begin(), h_input.end(), d_fluorescences);
    thrust::copy(h_bounds.begin(), h_bounds.end(), d_bounds);

    device::create_cells_from_fluorescence<<<n_blocks, n_threads_per_block>>>
        (n_threads_per_block, d_params, params_size, random_seed,
        h_input.size(), d_fluorescences,
        d_bounds,
        initial_size, d_cells);

    hipDeviceSynchronize();

    hipMemcpy(h_cells, d_cells, bytes, hipMemcpyDeviceToHost);

    hipFree(d_fluorescences);
    hipFree(d_bounds);
    hipFree(d_cells);
}

__host__
cell_type
create_cell_type(int32_t name, double_t probability,
                    double_t timer, double_t sigma)
{

    cell_type type =
    {
        .name = name,
        .probability = probability,
        .timer = timer,
        .sigma = sigma
    };

    return type;
}

namespace device
{

__global__
void
create_cells_from_fluorescence(uint64_t n_threads_per_block,
                                cell_type* d_params, uint64_t size,
                                uint64_t seed,
                                uint64_t groups_count, fluorescence* data,
                                uint64_t* bounds,
                                uint64_t initial_size, cell* d_cells)
{
    uint64_t id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < groups_count)
    {
        double_t f = data[id].value;
        uint64_t total = data[id].frequency;
        uint16_t n_blocks = round(0.5 + total / n_threads_per_block);

        device::create_cells_population<<<n_blocks, n_threads_per_block>>>
            (d_params, size, seed, total, bounds[id], d_cells, f);

    }
}

__global__
void
create_cells_population(cell_type* d_params, uint64_t size,
                        uint64_t seed, uint64_t initial_size,
                        uint64_t offset,
                        cell* d_cells, double_t fluorescence_value)
{
    uint64_t id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < initial_size)
    {
        seed = seed + id + fluorescence_value * 10000;

        cell c = create_cell(d_params, size, seed,
                            -1, fluorescence_value, 0);
        d_cells[id + offset] = c;
    }

}
    
} // End device namespace

} // End simulation namespace

} // End procell namespace
