#include "hip/hip_runtime.h"
#include <inttypes.h>
#include <math.h>
#include "simulation/cell.h"
#include "simulation/cells_population.h"

namespace procell { namespace simulation
{

void
create_cells_population(uint64_t initial_size, cell* h_cells)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0 /* TODO check devices number */);
    
    uint16_t n_threads_per_block = prop.maxThreadsPerBlock;
    uint16_t n_blocks = ceil(initial_size / n_threads_per_block);

    uint64_t bytes = initial_size * sizeof(cell);

    cell* d_cells;
    hipError_t err = hipMalloc((void**) &d_cells, bytes);
    
    device::create_cells_population<<<n_blocks, n_threads_per_block>>>
        (initial_size, d_cells);

    hipDeviceSynchronize();

    hipMemcpy(h_cells, d_cells, bytes, hipMemcpyDeviceToHost);

    hipFree(d_cells);
}

namespace device
{
    
__global__
void
create_cells_population(uint64_t n, cell* d_cells)
{
    uint64_t id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < n)
    {
        cell c;
        c.t = 0;
        d_cells[id] = c;
    }

}
    
} // End device namespace

} // End simulation namespace

} // End procell namespace
